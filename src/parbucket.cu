#include "hip/hip_runtime.h"
#include "parbucket.cuh"
#include "parDjikstra.h"
#include "utils.h"


namespace parheap{

template <class Ktype>
__global__
void BH_iter(ParBucketHeap<Ktype> bh,
		int inputSize, int numEdges, int numVertices,
		VoxBucketItem<Ktype>* srcNodes,
		int* distance,
		AdjacentNode* adjList,
		int* edgesOffset,
		int* edgesSize,
		bool* settled,
		bool* finished,
		Ktype destination)
{
	const int level=blockIdx.x;
	const int thid=threadIdx.x;

	// do update

	if(level==0)
	{
//		// insert all, initialize
		for(int iv=0;iv<numVertices;iv++) // iv: index of vertex
		{
			bool is_src=false;
			int init_priority=0;
			for(int is=0;is<inputSize;is++) // is: index of src
			{
				if(iv==srcNodes[is].key)
				{
					is_src=true;
					break;
				}
			}
			if(is_src)
			{
				init_priority=0;
			}else
			{
				init_priority=INT_MAX-1;
			}
			bh.updateRes(iv,init_priority);
			bh.d_mem[0]++; // cnt count
			distance[iv]=init_priority;
		}
		VoxBucketItem<Ktype> eOut;
		for(int round=0;round<numVertices;round++)
		{
			bh.extractMinRes(eOut);
			bh.d_mem[0]++; // cnt count
			// check finish
			if(eOut.key==destination)
			{
				*finished=true;
				break;
			}
			// mark v as settled
			Ktype v=eOut.key;
			int p=eOut.priority;
			settled[v]=1;

			for(int ie=edgesOffset[v];ie<edgesOffset[v]+edgesSize[v];ie++) // ie: index of edge
			{
				// if u is not settled, update((u,p+w))
				AdjacentNode v2u=adjList[ie];
				Ktype u=v2u.terminalVertex; // index of terminal vertex
				int w=v2u.weight;
				if(settled[u])
					continue;
				if(distance[u]>p+w)
				{
					distance[u]=p+w;
					bh.updateRes(u,p+w);
				}
			}
		}
	}
	else // level>0
	{
		do
		{
			int isFail3=0;
			//resolve
			isFail3=bh.Resolve(level);  // can fail because !metConstrain
		}while(!*finished);
	}
}



int parDijkstra(std::vector<int> &srcNode,
		Graph<AdjacentNode> &cuGraph,
		std::vector<int> &distances,
		int destination)
{
	using thrust::raw_pointer_cast;


	///  initCudaGraph
	int inputSize=srcNode.size();
	std::vector<VoxBucketItem<int>> h_srcNode(inputSize);
	thrust::device_vector<VoxBucketItem<int>> d_srcNode(inputSize);
	for(int id=0;id<inputSize;id++)
	{
		h_srcNode[id].setVal(srcNode[id],id%5);
	}

	thrust::copy(h_srcNode.begin(),h_srcNode.end(),d_srcNode.begin());

	thrust::device_vector<int> d_distance(cuGraph.numVertices);
	thrust::copy(distances.begin(),distances.end(),d_distance.begin());

	thrust::device_vector<AdjacentNode> d_adjLists(cuGraph.numEdges);
	thrust::device_vector<int> d_edgesOffset(cuGraph.numVertices);
	thrust::device_vector<int> d_edgesSize(cuGraph.numVertices);
	thrust::copy(cuGraph.adjacencyList.begin(),cuGraph.adjacencyList.end(),d_adjLists.begin());
	thrust::copy(cuGraph.edgesOffset.begin(),cuGraph.edgesOffset.end(),d_edgesOffset.begin());
	thrust::copy(cuGraph.edgesSize.begin(),cuGraph.edgesSize.end(),d_edgesSize.begin());

	thrust::device_vector<bool> d_settled(cuGraph.numVertices);


	///// Pardjikstra
	ParBucketHeap<int> bh(cuGraph.numVertices,1);
	int block_size=1;
	int grid_size=bh.max_levels;

	bool *finished;
	CUDA_ALLOC_DEV_MEM(&finished,sizeof(int));
	CUDA_DEV_MEMSET(finished,0,sizeof(int));
	int* d_destination;
	CUDA_ALLOC_DEV_MEM(&d_destination,sizeof(int));
	CUDA_MEMCPY_H2D(d_destination,&destination,sizeof(int));


	BH_iter<int><<<grid_size,block_size>>>(bh,
			inputSize, cuGraph.numEdges, cuGraph.numVertices,
			raw_pointer_cast(&d_srcNode[0]),
			raw_pointer_cast(&d_distance[0]),
			raw_pointer_cast(&d_adjLists[0]),
			raw_pointer_cast(&d_edgesOffset[0]),
			raw_pointer_cast(&d_edgesSize[0]),
			raw_pointer_cast(&d_settled[0]),
			 finished,
			destination);


	CUDA_FREE_DEV_MEM(d_destination);
	CUDA_FREE_DEV_MEM(finished);

	int dest_dist=d_distance[destination];
	std::cout<<"finaldist= "<<dest_dist<<std::endl;

	int total_rounds=bh.dbg_shared[0];
	std::cout<<"total rounds= "<<total_rounds<<std::endl;


	return 0;
}
}

